#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
// #include <c10/cuda/CUDAGuard.h>
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

namespace {
template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_gaussian_density(
    scalar_t x, scalar_t y,
    scalar_t mean_x, scalar_t mean_y,
    scalar_t std_x, scalar_t std_y) {

    const auto x_exp_term = (x - mean_x) / std_x;
    const auto y_exp_term = (y - mean_y) / std_y;

    const auto exp_term = __expf(-0.5 * (x_exp_term * x_exp_term + y_exp_term * y_exp_term));
    const auto std_term = 1.0 / (std_x * std_y);
    const auto const_term = 1.0 / (2.0 * 3.141592654);

    return exp_term * std_term * const_term;
}


template <typename scalar_t>
__device__ __forceinline__ scalar_t d_normal_pdf_d_mu_i(
    scalar_t x, scalar_t mean_x, scalar_t std_x, scalar_t density_value) {

    return density_value * (x - mean_x) / (std_x * std_x);
}


template <typename scalar_t>
__device__ __forceinline__ scalar_t d_normal_pdf_d_std_i(
    scalar_t x, scalar_t mean_x, scalar_t std_x, scalar_t density_value) {
    const auto d_denom_term = -density_value / std_x;
    const auto d_exp_term = density_value * (x - mean_x) * (x - mean_x) / (std_x * std_x * std_x);

    return d_denom_term + d_exp_term;
}


template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_distance_weight(
    scalar_t x, scalar_t y,
    scalar_t mean_x, scalar_t mean_y) {

    const auto x_diff = (x - mean_x);
    const auto y_diff = (y - mean_y);
    const auto dist2 = x_diff * x_diff + y_diff * y_diff;
    const auto weight = 1.0 / dist2;

    return weight;
}


template <typename scalar_t>
__device__ scalar_t clamp(scalar_t x, scalar_t min_val, scalar_t max_val) {
    return max(min_val, min(max_val, x));
}


template <typename scalar_t>
__device__ scalar_t compute_color_bilinear(
    scalar_t x,
    scalar_t y,
    const int c,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> image,
    bool use_d_x = false,
    bool use_d_y = false) {
    // Selects a color from an image using bilinear interpolation

    // i, j = math.floor(y.item()), math.floor(x.item())
    const int j = min(max(static_cast<int>(floor(x)), 0), image.size(2));
    const int i = min(max(static_cast<int>(floor(y)), 0), image.size(1));

    scalar_t c0 = image[c][i][j];
    scalar_t c1 = image[c][i][min(j + 1, image.size(2) - 1)];
    scalar_t c2 = image[c][min(i + 1, image.size(1) - 1)][min(j + 1, image.size(2) - 1)];
    scalar_t c3 = image[c][min(i + 1, image.size(1) - 1)][j];

    // Computing the derivative here is relatively easy
    // Since it is a linear function wrt to mu_x and mu_y,
    // We just replace them with ones in those places where they are used
    scalar_t w_top = use_d_y ? 1.0 : (y - static_cast<scalar_t>(i));
    scalar_t w_left = use_d_x ? 1.0 : (x - static_cast<scalar_t>(j));

    // Step 1: interpolate along x-axis
    scalar_t color_top = c0 * (1.0 - w_left) + c1 * w_left;
    scalar_t color_bot = c3 * (1.0 - w_left) + c2 * w_left;

    // Step 2: interpolate along y-axis
    scalar_t color = color_top * (1.0 - w_top) + w_top * color_bot;

    return color;
}



template <typename scalar_t>
__global__ void gp_interp_compute_color_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> image,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> means,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> stds,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> output_image,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> pixel_weights) {

    const int point_idx = blockDim.y * blockIdx.y + blockIdx.x;
    const int radius = blockDim.x / 2;
    const int center_x = static_cast<int>(clamp(round(means[point_idx][0]), static_cast<scalar_t>(0.0), static_cast<scalar_t>(image.size(2))));
    const int center_y = static_cast<int>(clamp(round(means[point_idx][1]), static_cast<scalar_t>(0.0), static_cast<scalar_t>(image.size(1))));
    const int shift_x = threadIdx.x - radius;
    const int shift_y = threadIdx.y - radius;
    const int pixel_pos_x = center_x + shift_x;
    const int pixel_pos_y = center_y + shift_y;

    if (pixel_pos_x >= 0 && pixel_pos_y >= 0 && pixel_pos_x < image.size(2) && pixel_pos_y < image.size(1)) {
        scalar_t weight = compute_gaussian_density(
            static_cast<scalar_t>(pixel_pos_x),
            static_cast<scalar_t>(pixel_pos_y),
            means[point_idx][0],
            means[point_idx][1],
            stds[point_idx][0],
            stds[point_idx][1]);
        // scalar_t weight = compute_distance_weight(
        //     static_cast<scalar_t>(pixel_pos_x),
        //     static_cast<scalar_t>(pixel_pos_y),
        //     means[point_idx][0],
        //     means[point_idx][1]);
        // scalar_t weight = means[point_idx][0] + means[point_idx][1];

        if (weight > 0.0) {
            for (int c = 0; c < image.size(0); c++) {
                // const scalar_t color_value = image[c][center_y][center_x]; // TODO: keep the color in the shared memory
                const scalar_t color_value = compute_color_bilinear(means[point_idx][0], means[point_idx][1], c, image);
                atomicAdd(&output_image[c][pixel_pos_y][pixel_pos_x], color_value * weight);
            }

            atomicAdd(&pixel_weights[pixel_pos_y][pixel_pos_x], weight);
        }
    }
}


template <typename scalar_t>
__global__ void gp_interp_normalize_color_kernel(
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> output_image,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> pixel_weights) {

    const int pixel_pos_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int pixel_pos_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (pixel_pos_x >= 0 && pixel_pos_y >= 0 && pixel_pos_x < output_image.size(2) && pixel_pos_y < output_image.size(1)) {
        const scalar_t weight = pixel_weights[pixel_pos_y][pixel_pos_x];

        if (weight > 0.0) {
            for (int c = 0; c < output_image.size(0); c++) {
                output_image[c][pixel_pos_y][pixel_pos_x] /= weight;
            }
        }
    }
}


template <typename scalar_t>
__global__ void gp_interp_cuda_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> grad_output_image,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> image,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> means,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> stds,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> output_image,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> pixel_weights,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> grad_image,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> grad_means,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> grad_stds) {

    const int point_idx = blockDim.y * blockIdx.y + blockIdx.x;
    const int radius = blockDim.x / 2;
    const int center_x = static_cast<int>(clamp(round(means[point_idx][0]), static_cast<scalar_t>(0.0), static_cast<scalar_t>(image.size(2))));
    const int center_y = static_cast<int>(clamp(round(means[point_idx][1]), static_cast<scalar_t>(0.0), static_cast<scalar_t>(image.size(1))));
    const int shift_x = threadIdx.x - radius;
    const int shift_y = threadIdx.y - radius;
    const int pixel_pos_x = center_x + shift_x;
    const int pixel_pos_y = center_y + shift_y;

    if (pixel_pos_x >= 0 && pixel_pos_y >= 0 && pixel_pos_x < image.size(2) && pixel_pos_y < image.size(1)) {
        scalar_t point_density = compute_gaussian_density(
            static_cast<scalar_t>(pixel_pos_x),
            static_cast<scalar_t>(pixel_pos_y),
            means[point_idx][0],
            means[point_idx][1],
            stds[point_idx][0],
            stds[point_idx][1]);
        scalar_t total_weight = pixel_weights[pixel_pos_y][pixel_pos_x];

        if (point_density > 0.0) {
            scalar_t d_v_d_mu_x = 0.0;
            scalar_t d_v_d_mu_y = 0.0;
            scalar_t d_v_d_std_x = 0.0;
            scalar_t d_v_d_std_y = 0.0;

            scalar_t common_multiplier = (1.0 - point_density / total_weight) / total_weight;
            // scalar_t common_multiplier = 1.0 / total_weight;
            scalar_t d_weight_d_mu_x = common_multiplier * d_normal_pdf_d_mu_i(
                static_cast<scalar_t>(pixel_pos_x), means[point_idx][0], stds[point_idx][0], point_density);
            scalar_t d_weight_d_mu_y = common_multiplier * d_normal_pdf_d_mu_i(
                static_cast<scalar_t>(pixel_pos_y), means[point_idx][1], stds[point_idx][1], point_density);
            scalar_t d_weight_d_std_x = common_multiplier * d_normal_pdf_d_std_i(
                static_cast<scalar_t>(pixel_pos_x), means[point_idx][0], stds[point_idx][0], point_density);
            scalar_t d_weight_d_std_y = common_multiplier * d_normal_pdf_d_std_i(
                static_cast<scalar_t>(pixel_pos_y), means[point_idx][1], stds[point_idx][1], point_density);

            for (int c = 0; c < image.size(0); c++) {
                const scalar_t d_loss_d_pixel_color = grad_output_image[c][pixel_pos_y][pixel_pos_x];
                // const scalar_t color_value = image[c][center_y][center_x]; // TODO: keep the color in the shared memory?
                const scalar_t color_value = compute_color_bilinear(means[point_idx][0], means[point_idx][1], c, image);
                const scalar_t d_color_d_mu_x = compute_color_bilinear(means[point_idx][0], means[point_idx][1], c, image, true, false);
                const scalar_t d_color_d_mu_y = compute_color_bilinear(means[point_idx][0], means[point_idx][1], c, image, false, true);

                d_v_d_mu_x += d_loss_d_pixel_color * (d_color_d_mu_x * point_density / total_weight + color_value * d_weight_d_mu_x);
                d_v_d_mu_y += d_loss_d_pixel_color * (d_color_d_mu_y * point_density / total_weight + color_value * d_weight_d_mu_y);
                d_v_d_std_x += d_loss_d_pixel_color * color_value * d_weight_d_std_x;
                d_v_d_std_y += d_loss_d_pixel_color * color_value * d_weight_d_std_y;
            }

            atomicAdd(&grad_means[point_idx][0], d_v_d_mu_x);
            atomicAdd(&grad_means[point_idx][1], d_v_d_mu_y);
            atomicAdd(&grad_stds[point_idx][0], d_v_d_std_x);
            atomicAdd(&grad_stds[point_idx][1], d_v_d_std_y);
        }
    }
}
} // namespace

std::vector<torch::Tensor> gp_interp_cuda_forward(
    torch::Tensor image,
    torch::Tensor means,
    torch::Tensor stds,
    int radius) {

    const auto num_points = means.size(0);
    auto output_image = torch::zeros_like(image).contiguous();
    auto pixel_weights = torch::zeros({image.size(1), image.size(2)}).to(output_image.device()).contiguous();

    {
        const dim3 threads(radius * 2 + 1, radius * 2 + 1);
        const int blocks = num_points;

        AT_DISPATCH_FLOATING_TYPES(image.type(), "gp_interp_compute_color_kernel", ([&] {
        gp_interp_compute_color_kernel<scalar_t><<<blocks, threads>>>(
            image.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            means.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
            stds.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
            output_image.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            pixel_weights.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
        }));
        AT_CUDA_CHECK(hipGetLastError());
    }

    {
        const dim3 threads(16, 16);
        const dim3 blocks((image.size(2) + 16 - 1) / 16, (image.size(1) + 16 - 1) / 16);
        AT_DISPATCH_FLOATING_TYPES(image.type(), "gp_interp_normalize_color_kernel", ([&] {
        gp_interp_normalize_color_kernel<scalar_t><<<blocks, threads>>>(
            output_image.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            pixel_weights.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
        }));
        AT_CUDA_CHECK(hipGetLastError());
    }

    return {output_image, pixel_weights};
}


std::vector<torch::Tensor> gp_interp_cuda_backward(
    torch::Tensor grad_output_image,
    torch::Tensor image,
    torch::Tensor means,
    torch::Tensor stds,
    int radius,
    torch::Tensor output_image,
    torch::Tensor pixel_weights) {

    auto grad_image = torch::zeros_like(image).contiguous();
    auto grad_means = torch::zeros_like(means).contiguous();
    auto grad_stds = torch::zeros_like(stds).contiguous();

    const dim3 threads(radius * 2 + 1, radius * 2 + 1);
    const auto num_points = means.size(0);
    const int blocks = num_points;

    AT_DISPATCH_FLOATING_TYPES(image.type(), "gp_interp_cuda_backward", ([&] {
    gp_interp_cuda_backward_kernel<scalar_t><<<blocks, threads>>>(
        grad_output_image.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
        image.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
        means.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
        stds.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
        output_image.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
        pixel_weights.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
        grad_image.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
        grad_means.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
        grad_stds.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
    }));
    AT_CUDA_CHECK(hipGetLastError());

    return {grad_image, grad_means, grad_stds};
}
